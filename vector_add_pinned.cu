
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <hip/hip_runtime_api.h>

#include <stdio.h>

#define N 200000000

__global__ void vector_add(float *out, float *a, float *b){
    int i = threadIdx.x;
    out[i] = a[i] + b[i];
}

int run_one_iter(int current_iter){
    float *a, *b, *out;
    // allocate PINNED memory on host
    if(current_iter == 0){
        hipError_t err1 = hipHostMalloc((void**)&a, sizeof(float)*N, hipHostMallocDefault);
        hipError_t err2 = hipHostMalloc((void**)&b, sizeof(float)*N, hipHostMallocDefault);
        hipError_t err3 = hipHostMalloc((void**)&out, sizeof(float)*N, hipHostMallocDefault);
    }
   
    // initialize arrays
    for(int i=0; i<N; i++){
        a[i] = i * 1.0f;
        b[i] = i * 2.0f;
    }

    float *d_a, *d_b, *d_out;
    // allocate memory on device
    hipMalloc((void**)&d_a, sizeof(float)*N);
    hipMalloc((void**)&d_b, sizeof(float)*N);
    hipMalloc((void**)&d_out, sizeof(float)*N);

    // transfert data to it
    hipEvent_t start_cpy, end_cpy, start_kernel, end_kernel;
    hipEventCreate(&start_cpy);
    hipEventCreate(&end_cpy);
    hipEventCreate(&start_kernel);
    hipEventCreate(&end_kernel);

    hipEventRecord(start_cpy);
    hipMemcpy(d_a, a, sizeof(float)*N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float)*N, hipMemcpyHostToDevice);
    hipEventRecord(end_cpy);
    hipEventSynchronize(end_cpy);
    float millisec = 0;
    hipEventElapsedTime(&millisec, start_cpy, end_cpy);
    printf("Copy execution time: %f ms\n", millisec);
    hipEventDestroy(start_cpy); 
    hipEventDestroy(end_cpy); 
    // Nvidia 960M has 5 SM, 32 blocks per SM, 2064 max threads per SM
    // so we use all blocks with 64 threads per block
    // warp size is 32, hence the number of threads

    hipEventRecord(start_kernel);
    vector_add<<<160,64>>>(d_out, d_a, d_b);
    hipEventRecord(end_kernel);
    hipEventSynchronize(end_kernel);
    float millisec_k = 0;
    hipEventElapsedTime(&millisec_k, start_kernel, end_kernel);
    printf("Kernel execution time: %f ms\n", millisec_k);
    hipEventDestroy(start_kernel); 
    hipEventDestroy(end_kernel); 

    hipMemcpy(out, d_out, sizeof(float)*N, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
    if(current_iter == 9){
        hipHostFree(a);
        hipHostFree(b);
        hipHostFree(out);
    }
   return 0;    
}

int main(){
    int n = 10;
    for(int i=0; i<n; i++){
        run_one_iter(i);
    }
    return 0;
}